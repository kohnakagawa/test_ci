
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

__global__ void func(int* a) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  a[tid] = tid;
}

void check(const int* a,
           const int size) {
  for (int i = 0; i < size; i++) {
    if (a[i] != i) {
      std::cerr << "Error occurs (a[i] != i) at " << __FILE__ << " " << __LINE__ << "\n";
      std::exit(1);
    }
  }
}

int main() {
  const int tb_size = 128;
  const int gr_size = 1000;
  const auto size = tb_size * gr_size;

  int* a_h = new int[size];
  int* a_d = nullptr;
  hipMalloc((void**)&a_d, size * sizeof(int));

  func<<<gr_size, tb_size>>>(a_d);
  hipMemcpy(a_h, a_d,
             size * sizeof(int),
             hipMemcpyDeviceToHost);
  check(a_h, size);
  hipFree(a_d);
  delete [] a_h;
}
